#include <array>
#include <cstddef>
#include <string>

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>

#include "mex.h"

#include "gpu/mxGPUArray.h"

namespace
{
  void iMakeInterleaved(void const* const inPtr, void*& outPtr, std::size_t const batchSize,
                        std::size_t const numBatches, mxClassID const type)
  {
    std::size_t const elemBytes = (type == mxSINGLE_CLASS) ? sizeof(float) : sizeof(double);
    auto const cudaStatus = hipMalloc((void**)&outPtr, elemBytes * batchSize * numBatches);

    if (cudaStatus != hipSuccess)
    {
      char constexpr errId[] = "parallel:gpu:bandedSolveGPUmex:MallocError";
      char constexpr errMsg[] = "Unable to allocate GPU memory";
      mexErrMsgIdAndTxt(errId, errMsg);
    }

    float constexpr one = 1;
    float constexpr zero = 0;
    hipblasHandle_t cublasH = NULL;

    auto hipblasStatus_t = hipblasCreate(&cublasH);
    if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS)
    {
      char constexpr errId[] = "parallel:gpu:bandedSolveGPUmex:InitError";
      std::string const errMsg = "Unable to init cublas: Error ";
      mexErrMsgIdAndTxt(errId, (errMsg + std::to_string(hipblasStatus_t)).c_str());
    }

    hipblasStatus_t = hipblasSgeam(cublasH, HIPBLAS_OP_T,    /* transa */
                               HIPBLAS_OP_T,             /* transb, don't care */
                               batchSize,               /* number of rows of ds */
                               numBatches,              /* number of columns of ds */
                               &one, (float*)inPtr,     /* ds0 is n-by-batchSize */
                               numBatches,              /* leading dimension of ds0 */
                               &zero, NULL, numBatches, /* don't cae */
                               (float*)outPtr,          /* ds is batchSize-by-n */
                               batchSize);              /* leading dimension of ds */

    if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS)
    {
      char constexpr errId[] = "parallel:gpu:bandedSolveGPUmex:InterleaveError";
      std::string const errMsg = "Unable to interleave (cublasXgeam): Error ";
      mexErrMsgIdAndTxt(errId, (errMsg + std::to_string(hipblasStatus_t)).c_str());
    }
  };

} // namespace

// Penta-Diagonal Banded Solver
void mexFunction(int nlhs, mxArray* plhs[], int nrhs, mxArray const* prhs[])
{
  std::size_t constexpr N = 5;          // Number of diagonals
  std::size_t constexpr inArgs = N + 1; // Number of inputs

  // Check number of inputs
  if (nrhs != inArgs)
  {
    char constexpr errId[] = "parallel:gpu:bandedSolveGPUmex:InvalidInputs";
    char constexpr errMsg[] = "Invalid number of inputs.";
    mexErrMsgIdAndTxt(errId, errMsg);
  }

  // Ensure everything is a GPUarray
  for (auto i = 0; i < nrhs; i++)
    if (!mxIsGPUArray(prhs[i]))
    {
      char constexpr errId[] = "parallel:gpu:bandedSolveGPUmex:NotOnGPU";
      std::string const errMsg = "Input not a GPU array: ";
      mexErrMsgIdAndTxt(errId, (errMsg + std::to_string(i)).c_str());
    }

  /* Initialize the MathWorks GPU API. */
  mxInitGPU();

  // Extract GPUArrays from all inputs
  std::array<mxGPUArray const* const, 5> const bands = {mxGPUCreateFromMxArray(prhs[0]),  // minus 2
                                                        mxGPUCreateFromMxArray(prhs[1]),  // minus 1
                                                        mxGPUCreateFromMxArray(prhs[2]),  // main
                                                        mxGPUCreateFromMxArray(prhs[3]),  // plus 1
                                                        mxGPUCreateFromMxArray(prhs[4])}; // plus 2
  auto const Y = mxGPUCreateFromMxArray(prhs[5]);                                         // RHS

  // Ensure everything is the same data type
  auto const dataType = mxGPUGetClassID(Y);
  for (auto i = 0; i < N; i++)
  {
    if (mxGPUGetClassID(bands[i]) != dataType)
    {
      char constexpr errId[] = "parallel:gpu:bandedSolveGPUmex:InconsistentType";
      char constexpr errMsg[] = "Inputs are of different types";
      mexErrMsgIdAndTxt(errId, errMsg);
    }
  }

  // Get raw pointers
  std::array<void const*, 5> const bandsPtr = {mxGPUGetDataReadOnly(bands[0]),  // minus 2
                                               mxGPUGetDataReadOnly(bands[1]),  // minus 1
                                               mxGPUGetDataReadOnly(bands[2]),  // main
                                               mxGPUGetDataReadOnly(bands[3]),  // plus 1
                                               mxGPUGetDataReadOnly(bands[4])}; // plus 2
  void const* const YPtr = mxGPUGetDataReadOnly(Y);                             // RHS

  // Step 1: Convert to Interleave
  std::array<void*, 5> bandsInterleavedPtr;
  void* YInterleavedPtr;

  // Total elements
  auto const dims = mxGPUGetDimensions(Y);
  auto const batchesSize = dims[0];
  auto const numBatches = dims[1];
  mxFree((void*)dims);

  for (auto i = 0; i < N; i++)
  {
    iMakeInterleaved(bandsPtr[i], bandsInterleavedPtr[i], batchesSize, numBatches, dataType);
    mxGPUDestroyGPUArray(bands[i]);
  }

  iMakeInterleaved(YPtr, YInterleavedPtr, batchesSize, numBatches, dataType);
  mxGPUDestroyGPUArray(Y);

  for (auto i = 0; i < N; i++)
  {
    hipFree(bandsInterleavedPtr[i]);
  }
  hipFree(YInterleavedPtr);

  //   for (auto i = 0; i < nrhs)
  //     auto const ds = ;
  //   auto const dl = mxGPUCreateFromMxArray(prhs[0]);
  //   auto const ds = mxGPUCreateFromMxArray(prhs[0]);
  //   auto const ds = mxGPUCreateFromMxArray(prhs[0]);
  //   auto const ds = mxGPUCreateFromMxArray(prhs[0]);
  //   auto const ds = mxGPUCreateFromMxArray(prhs[0]);

  //   /* Declare all variables.*/
  //   mxGPUArray const* A;
  //   mxGPUArray* B;
  //   double const* d_A;
  //   double* d_B;
  //   int N;

  //   /* Choose a reasonably sized number of threads for the block. */
  //   int const threadsPerBlock = 256;
  //   int blocksPerGrid;
}
